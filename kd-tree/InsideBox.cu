#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>
#include "InsideBox.hpp"


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);


//think about: threadIDx.y!! index s.d. threadIDx.y die treeArray_y bearbeitet -
//for this would have to change treeArray_x & treeArray_y etc --> tree..
__global__
void insideBox( int *treeArray_x, int *treeArray_y, int *treeArray_z, int *treeArray_ID, int *box)
{
    //braucht kein d_result.
    //Kann in treeArray_ID returned werden -1, wenn nicht insideBox
    int warpSize = 32;
    int warpIdx = threadIdx.x / warpSize;
    int i = warpIdx;
    int j = threadIdx.x % 32; //=0 bis 32
    int treeSize = warpSize-1;
    int index = i*treeSize+j;
    
    if( ((treeArray_x[index] >= box[0] && treeArray_x[index] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[index] >= box[2] && treeArray_y[index] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[index] >= box[4] && treeArray_z[index] <= box[5]) || (box[4] == 0 && box[5] == 0))){
        //inside box
        
    }
    else{
        treeArray_ID[index] = -1;
    }
}

void cudaMain(int number_of_trees, int tree_size, int treeArray_x[], int treeArray_y[], int treeArray_z[], int treeArray_ID[], int box[]){
    
    hipSetDevice(MYDEVICE);
    
    
    //TODO: int ----> num_t
    int size_of_forest = number_of_trees*tree_size*sizeof(int);
    int *d_treeArray_x;
    int *d_treeArray_y;
    int *d_treeArray_z;
    int *d_treeArray_ID;
    int *d_box;
    //int size_of_forest = sizeof(int)*trees.size()*trees[0].size();
    
    
    //allocate memory
    hipMalloc(&d_treeArray_x, size_of_forest);
    hipMalloc(&d_treeArray_y, size_of_forest);
    hipMalloc(&d_treeArray_z, size_of_forest);
    hipMalloc(&d_treeArray_ID, size_of_forest);
    //TODO: generic
    hipMalloc(&d_box, 6*sizeof(int));
    
    //send trees to gpu
    hipMemcpy(d_treeArray_x, treeArray_x, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_y, treeArray_y, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_z, treeArray_z, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_ID, treeArray_ID, size_of_forest, hipMemcpyHostToDevice);
    //TODO: generic
    hipMemcpy(d_box, box, 6*sizeof(int), hipMemcpyHostToDevice);
    //here we have to split the forest? - split on GPU?
    
    //TODO:kernel, s.d. jeder einzelne thread checkt, ob in box - box-dimensionen gegeben
    //gibt zurück ein array mit punkten, die in box (coordinaten? ID's? .. )
    //main.cpp -> main.cu und andere compilation von c++11 zeug muss ausgelagert werden
    //search forest for points inside box_dimensions
    
    insideBox<<<1,1>>>(d_treeArray_x, d_treeArray_y, d_treeArray_z, d_treeArray_ID, d_box);
    
    //DO NOT NEED - USE treeArray_ID
    //allocate host and device memory for results - ID's of hits/datapoints inside box
    /*int* h_result;
    int* d_result;
    size_t resultSize = numberOfHits*sizeof(int);
    h_result = (int *) malloc(resultSize);
    hipMalloc((void **)&d_result, resultSize);
    */
    hipMemcpy(treeArray_ID, d_treeArray_ID, size_of_forest, hipMemcpyDeviceToHost);
    
    //TODO: print out ID's which were in box:
    
    
    
    //free space
    hipFree(d_treeArray_x);
    hipFree(d_treeArray_y);
    hipFree(d_treeArray_z);
    hipFree(d_treeArray_ID);
    hipFree(d_box);
    /*free(treeArray_x);
    free(treeArray_y);
    free(treeArray_z);
    free(treeArray_ID);
    free(box);*/

}
