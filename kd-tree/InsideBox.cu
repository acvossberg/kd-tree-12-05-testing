#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);



__global__ void testKernel( int *trees, int numBlocks )
{
    int i = numBlocks*blockIdx.x + threadIdx.x;
    d_a[i] = blockIdx.x + threadIdx.x;
    
}

void cudaMain(){
    hipSetDevice(MYDEVICE);



}
