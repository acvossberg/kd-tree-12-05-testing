#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>
#include <iostream>
#include "InsideBox.hpp"


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);


//think about: threadIDx.y!! index s.d. threadIDx.y die treeArray_y bearbeitet -
//for this would have to change treeArray_x & treeArray_y etc --> tree..
//no nested if's no recursive.. -
template <typename T>
__global__
void insideBox_test( T *treeArray_x, T *treeArray_y, T *treeArray_z, int *treeArray_ID, T *box)
{
    //TODO: change to get warpSize
    int warpSize = 32;
    int warpIdx = threadIdx.x / warpSize;
    int i = warpIdx;
    int j = threadIdx.x % 32; //=0 bis 32
    int treeSize = warpSize-1;
    int index = i*treeSize+j;
    
    
    
    //evaluate with another if(which is the next, then you exit from the if.. and you follow the next
    if( ((treeArray_x[index] >= box[0] && treeArray_x[index] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[index] >= box[2] && treeArray_y[index] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[index] >= box[4] && treeArray_z[index] <= box[5]) || (box[4] == 0 && box[5] == 0))){
        //inside box
        
    }
    else{
        //printf("not inside box");
        treeArray_ID[index] = -1;
    }
}

template <typename T>
__device__
void traverseTree( T *treeArray_x, T *treeArray_y, T *treeArray_z, int *treeArray_ID, T *box, int pos, int startOfTree, int endOfTree){
    
    //printf("\n threadIdx: %d startOfTree %d, endOfTree %d", threadIdx.x, startOfTree, endOfTree);
    
    if(startOfTree + pos -1 <= endOfTree){
    
        ///CHECK HERE!!!! STARTOFTREE+POS == INDEX  AND POS = i , i->2*i etc.
        if( ((treeArray_x[startOfTree+pos] >= box[0] && treeArray_x[startOfTree+pos] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[startOfTree+pos] >= box[2] && treeArray_y[startOfTree+pos] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[startOfTree+pos] >= box[4] && treeArray_z[startOfTree+pos] <= box[5]) || (box[4] == 0 && box[5] == 0))){
            //inside box
        }
        else{
            //printf("\n not inside box at position %d with thread nr: %d ", startOfTree+pos, threadIdx.x);
            treeArray_ID[startOfTree+pos] = -1;
        }
    
        //Abbruchkriterium:
        //TODO: < oder <= ???
        //if(startOfTree+pos < endOfTree){
        
            //left child:
            pos *= 2;
            traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, pos, startOfTree, endOfTree);
        
            //right child:
            pos += 1;
            traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, pos, startOfTree, endOfTree);
        //}
    }
}


//Each thread starts at Node 0 of it's "own" tree. Traverses tree and changes treeArray_ID to -1, if not inside box.
template <typename T>
__global__
void insideBox(T *treeArray_x, T *treeArray_y, T *treeArray_z, int *treeArray_ID, T *box, int tree_size){
    
    //for each thread has it's own tree starting here
    int startOfTree = threadIdx.x * tree_size ;
    int endOfTree = startOfTree + tree_size - 1;
    traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, 1, startOfTree, endOfTree);
}



//more generic approach!!

template <typename T>
__device__
void traverseTree( T *treeArray_values, int *treeArray_ID, T *box, int pos, int startOfTree, int endOfTree){
    
    //printf("\n threadIdx: %d startOfTree %d, endOfTree %d", threadIdx.x, startOfTree, endOfTree);
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(startOfTree + pos -1 <= endOfTree){
        ///CHECK HERE!!!! STARTOFTREE+POS == INDEX  AND POS = i , i->2*i etc.
       if( ((treeArray_values[row][startOfTree+pos] >= box[row] && treeArray_values[row][startOfTree+pos] <= box[row]) || (box[row] == 0 && box[row] == 0))  && ((treeArray_values[row][startOfTree+pos] >= box[row] && treeArray_values[row][startOfTree+pos] <= box[row]) || (box[row] == 0 && box[row] == 0)) && ((treeArray_values[row][startOfTree+pos] >= box[row] && treeArray_values[row][startOfTree+pos] <= box[row]) || (box[row] == 0 && box[row] == 0))){
            //inside box
        }
        else{
            //printf("\n not inside box at position %d with thread nr: %d ", startOfTree+pos, threadIdx.x);
            treeArray_ID[startOfTree+pos] = -1;
        }
        
        //Abbruchkriterium:
        //TODO: < oder <= ???
        //TODO: if hier weg
        if(startOfTree+pos < endOfTree){
            
            //left child:
            pos *= 2;
            traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree);
            
            //right child:
            pos += 1;
            traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree);
        }
    }
}

template <typename T>
__global__
void insideBox(T *treeArray_values, int *treeArray_ID, T *box, int tree_size, int number_of_dimensions){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    //for each thread has it's own tree starting here
    int startOfTree = threadIdx.x * tree_size ;
    int endOfTree = startOfTree + tree_size - 1;
    printf("\n threadIdx: %d startOfTree %d, endOfTree %d, row %d, col %d", threadIdx.x, startOfTree, endOfTree, row, col);
    //traverseTree(treeArray_values, treeArray_ID, box, 1, startOfTree, endOfTree);
}


template <typename T>
//void Cuda_class<T>::cudaMain(int number_of_trees, int tree_size, T treeArray_x[], T treeArray_y[], T treeArray_z[], int treeArray_ID[], T box[]){
void Cuda_class<T>::cudaMain(int number_of_trees, int tree_size, T *treeArray_values, int *treeArray_ID, T box[],  int number_of_dimensions){
    
    hipSetDevice(MYDEVICE);
    std::cout << "number of trees: " << number_of_trees << std::endl;
    std::cout << "tree size: " << tree_size << std::endl;
    std::cout << "number of dimensions: " << number_of_dimensions << std::endl;
    
    //TODO: int ----> num_t
    int size_of_forest = number_of_trees*tree_size*sizeof(int);
    T *d_treeArray_values;
    //T *d_treeArray_x;
    //T *d_treeArray_y;
    //T *d_treeArray_z;
    int *d_treeArray_ID;
    T *d_box;
    
    
    //allocate memory
    /*hipMalloc(&d_treeArray_x, size_of_forest);
    hipMalloc(&d_treeArray_y, size_of_forest);
    hipMalloc(&d_treeArray_z, size_of_forest);*/
    hipMalloc(&d_treeArray_values, size_of_forest);
    hipMalloc(&d_treeArray_ID, size_of_forest);
    //TODO: generic
    hipMalloc(&d_box, number_of_dimensions*2*sizeof(T));
    
    //send trees to gpu
    /*hipMemcpy(d_treeArray_x, treeArray_x, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_y, treeArray_y, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_z, treeArray_z, size_of_forest, hipMemcpyHostToDevice);*/
    hipMemcpy(d_treeArray_values, treeArray_values, size_of_forest*number_of_dimensions, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_ID, treeArray_ID, size_of_forest, hipMemcpyHostToDevice);
    //TODO: generic
    hipMemcpy(d_box, box, number_of_dimensions*2*sizeof(T), hipMemcpyHostToDevice);
    
    
    //search forest for points inside box_dimensions - returns all treeArray_ID's which are inside box - rest are filled with -1
    //TODO: do not change treeArray_ID's - make separate array.
    dim3 dimBlock(number_of_trees, number_of_dimensions);
    insideBox<T><<<1,dimBlock>>>(d_treeArray_values, d_treeArray_ID, d_box, tree_size, number_of_dimensions);
    //YourKernel<<<dimGrid, dimBlock>>>(d_A,d_B); //Kernel invocation
    
    /*
    //test wether insideBox works
    int *d_treeArray_ID_copy;
    int test_ID[number_of_trees*tree_size];
    //int test_treeArray_ID = std::copy(treeArray_ID);
    hipMalloc(&d_treeArray_ID_copy, size_of_forest);
    hipMemcpy(d_treeArray_ID_copy, treeArray_ID, size_of_forest, hipMemcpyHostToDevice);
    //insideBox_test<<<1,1024>>>(d_treeArray_x, d_treeArray_y, d_treeArray_z, d_treeArray_ID_copy, d_box);
    hipMemcpy(test_ID, d_treeArray_ID_copy, size_of_forest, hipMemcpyDeviceToHost);
    //finish test
     */
    
    hipMemcpy(treeArray_ID, d_treeArray_ID, size_of_forest, hipMemcpyDeviceToHost);
    
    /*
    bool correctID=true;
    for(int i = 0; i<number_of_trees*tree_size; i++){
        correctID = correctID && (treeArray_ID[i] == test_ID[i]);
    }
    printf("\n All ID's found in box are %d", correctID );
     */
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    
    std::cout << "\n Size of forest: " << size_of_forest << std::endl;
//
//    //print out ID's which are in box:
//    for(int i = 0; i< number_of_trees*tree_size; i++){
//        std::cout << "ID: " << treeArray_ID[i]<< std::endl;
//    }
    
    
    //free space
    /*hipFree(d_treeArray_x);
    hipFree(d_treeArray_y);
    hipFree(d_treeArray_z);*/
    hipFree(d_treeArray_values);
    hipFree(d_treeArray_ID);
    hipFree(d_box);

}
template class Cuda_class<int>;
