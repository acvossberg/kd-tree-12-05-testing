#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>
#include <iostream>
#include "InsideBox.hpp"


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);


//think about: threadIDx.y!! index s.d. threadIDx.y die treeArray_y bearbeitet -
//for this would have to change treeArray_x & treeArray_y etc --> tree..
//no nested if's no recursive.. -
template <typename T>
__global__
void insideBox_test( T *treeArray_x, T *treeArray_y, T *treeArray_z, int *treeArray_ID, T *box)
{
    //TODO: change to get warpSize
    int warpSize = 32;
    int warpIdx = threadIdx.x / warpSize;
    int i = warpIdx;
    int j = threadIdx.x % 32; //=0 bis 32
    int treeSize = warpSize-1;
    int index = i*treeSize+j;
    
    
    
    //evaluate with another if(which is the next, then you exit from the if.. and you follow the next
    if( ((treeArray_x[index] >= box[0] && treeArray_x[index] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[index] >= box[2] && treeArray_y[index] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[index] >= box[4] && treeArray_z[index] <= box[5]) || (box[4] == 0 && box[5] == 0))){
        //inside box
        
    }
    else{
        //printf("not inside box");
        treeArray_ID[index] = -1;
    }
}

template <typename T>
__device__
void traverseTree( T *treeArray_x, T *treeArray_y, T *treeArray_z, int *treeArray_ID, T *box, int pos, int startOfTree, int endOfTree){
    
    //printf("\n threadIdx: %d startOfTree %d, endOfTree %d", threadIdx.x, startOfTree, endOfTree);
    
    if(startOfTree + pos -1 <= endOfTree){
    
        ///CHECK HERE!!!! STARTOFTREE+POS == INDEX  AND POS = i , i->2*i etc.
        if( ((treeArray_x[startOfTree+pos] >= box[0] && treeArray_x[startOfTree+pos] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[startOfTree+pos] >= box[2] && treeArray_y[startOfTree+pos] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[startOfTree+pos] >= box[4] && treeArray_z[startOfTree+pos] <= box[5]) || (box[4] == 0 && box[5] == 0))){
            //inside box
        }
        else{
            //printf("\n not inside box at position %d with thread nr: %d ", startOfTree+pos, threadIdx.x);
            treeArray_ID[startOfTree+pos] = -1;
        }
    
        //Abbruchkriterium:
        //TODO: < oder <= ???
        //if(startOfTree+pos < endOfTree){
        
            //left child:
            pos *= 2;
            traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, pos, startOfTree, endOfTree);
        
            //right child:
            pos += 1;
            traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, pos, startOfTree, endOfTree);
        //}
    }
}


//Each thread starts at Node 0 of it's "own" tree. Traverses tree and changes treeArray_ID to -1, if not inside box.
template <typename T>
__global__
void insideBox(T *treeArray_x, T *treeArray_y, T *treeArray_z, int *treeArray_ID, T *box, int tree_size){
    
    //for each thread has it's own tree starting here
    int startOfTree = threadIdx.x * tree_size ;
    int endOfTree = startOfTree + tree_size - 1;
    traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, 1, startOfTree, endOfTree);
}



//more generic approach!!

template <typename T>
__device__
void traverseTree( T *treeArray_values, int *treeArray_ID, T *box, int pos, int startOfTree, int endOfTree, int number_of_dimensions){
    
    //int row = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("\n first value: startOfTree+pos*blokDim.y + row %d, row %d", (startOfTree+pos)*blockDim.y+row, row);
    //printf("\n box[0] %d, box[1] %d, box[2] %d, box[3] %d, box[4] %d, box[5] %d, thread %d", box[0], box[1], box[2], box[3], box[4], box[5], threadIdx.x);
    
    //TODO: erstes/ erst hits in tree wird nicht processed
    if(startOfTree + pos - 1 <= endOfTree){
        
        for( int i=0; i<number_of_dimensions; i++){
            if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                //inside box
                //printf("\n value: %d , box-min: %d, box-max: %d, ID: %d, thread %d", treeArray_values[startOfTree+number_of_dimensions*(pos-1)+i], box[2*i], box[2*i+1], treeArray_ID[startOfTree+pos-1], threadIdx.x);
            }
            else{
                //printf("\n value: %d , box-min: %d, box-max: %d, ID: %d, thread %d NOOOOOT", treeArray_values[startOfTree+number_of_dimensions*(pos-1)+i], box[2*i], box[2*i+1], treeArray_ID[startOfTree+pos-1], threadIdx.x);
                treeArray_ID[startOfTree+pos-1] = -1;
            }
            
        }
        
        //Abbruchkriterium:
        //TODO: < oder <= ???
        //TODO: if hier weg
        if(startOfTree+pos < endOfTree){
            
            //left child:
            pos *= 2;
            traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
            
            //right child:
            pos += 1;
            traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
        }
    }
}


template <typename T>
__global__
void insideBox(T *treeArray_values, int *treeArray_ID, T *box, int tree_size, int number_of_dimensions){
    
    //for each thread has it's own tree starting here
    //TODO: STARTOFTREE falsch.. ist die gesamte position, ohne berücksichtigung der number_of_dimensions. Die müssen berücksichtigt werden!!!
    int startOfTree = threadIdx.x * tree_size;
    int endOfTree = startOfTree + (tree_size - 1);
    for(int i = startOfTree; i<endOfTree; i++ ){
        //printf("\n (%d, %d, %d) \t ID: %d,\t startOfTree: %d, \t position of point: %d, thread: %d", treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*i+0], treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*i+1], treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*i+2], treeArray_ID[startOfTree+i], startOfTree*number_of_dimensions, startOfTree*number_of_dimensions+number_of_dimensions*i, threadIdx.x);
    }
    //printf("\n threadIdx: %d startOfTree %d, endOfTree %d, row %d, col %d, blockDim %d", threadIdx.x, startOfTree, endOfTree, row, col, blockDim.y);
    traverseTree(treeArray_values, treeArray_ID, box, 1, startOfTree, endOfTree, number_of_dimensions);
}


template <typename T>
//void Cuda_class<T>::cudaMain(int number_of_trees, int tree_size, T treeArray_x[], T treeArray_y[], T treeArray_z[], int treeArray_ID[], T box[]){
void Cuda_class<T>::cudaMain(int number_of_trees, int tree_size, T *treeArray_values, int *treeArray_ID, T box[],  int number_of_dimensions){
    
    hipSetDevice(MYDEVICE);
    std::cout << "number of trees: " << number_of_trees << std::endl;
    std::cout << "tree size: " << tree_size << std::endl;
    std::cout << "number of dimensions: " << number_of_dimensions << std::endl;
    std::cout << "box: " << box[0] << " " << box[1] << " " << box[2] << " " << box[3] << " " << box[4] << " " << box[5] << std::endl;
    //TODO: int ----> num_t
    int size_of_forest = number_of_trees*tree_size*sizeof(int);
    T *d_treeArray_values;
    int *d_treeArray_ID;
    T *d_box;
    
    
    //allocate memory
    //TODO: do outside of cudaMain
    hipMalloc(&d_treeArray_values, size_of_forest*number_of_dimensions);
    hipMalloc(&d_treeArray_ID, size_of_forest);
    hipMalloc(&d_box, number_of_dimensions*2*sizeof(T));
    
    //send trees to gpu
    hipMemcpy(d_treeArray_values, treeArray_values, size_of_forest*number_of_dimensions, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_ID, treeArray_ID, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_box, box, number_of_dimensions*2*sizeof(T), hipMemcpyHostToDevice);
    
    
    //search forest for points inside box_dimensions - returns all treeArray_ID's which are inside box - rest are filled with -1
    //TODO: do not change treeArray_ID's - make separate array.
    
    //insideBox<T><<<Anzahl benutzte Blöcke, Anzahl Threads>>> = <<<Anzahl benutzte Blöcke, Anzahl Baeume >>>
    //weil ein Thread == ein Baum
    insideBox<T><<<1,number_of_trees>>>(d_treeArray_values, d_treeArray_ID, d_box, tree_size, number_of_dimensions);
    //YourKernel<<<dimGrid, dimBlock>>>(d_A,d_B); //Kernel invocation
    
    
    hipMemcpy(treeArray_ID, d_treeArray_ID, size_of_forest, hipMemcpyDeviceToHost);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    
    std::cout << "\n Size of forest: " << size_of_forest << std::endl;

    //print out ID's which are in box:
//    for(int i = 0; i< number_of_trees*tree_size; i++){
//        std::cout << "ID: " << treeArray_ID[i]<< std::endl;
//    }
    
    
    //free space
    hipFree(d_treeArray_values);
    hipFree(d_treeArray_ID);
    hipFree(d_box);

}

template class Cuda_class<int>;
