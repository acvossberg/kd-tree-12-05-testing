#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);



__global__ void insideBox( int *trees, int numBlocks )
{
    int warpSize = 32;
    int warpIdx = threadIdx.x / warpSize;
    int i = warpIdx;
    int j = threadIdx.x % 32; //=0 bis 32
    
    if( ((trees[i][j].x >= start.x && trees[i][j].x <= end.x) || (end.x == 0 && start.x == 0))  && ((trees[i][j].y >= start.y && trees[i][j].y <= end.y) || (end.y == 0 && start.y == 0)) && ((trees[i][j].z >= start.z && trees[i][j].z <= end.z) || (end.z == 0 && start.z == 0))){
    }
    
    
    
    int i = numBlocks*blockIdx.x + threadIdx.x;
    d_a[i] = blockIdx.x + threadIdx.x;
    
}

void cudaMain(){
    hipSetDevice(MYDEVICE);



}
