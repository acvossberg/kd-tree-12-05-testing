#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>
#include <iostream>
#include "InsideBox.hpp"


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

//more generic approach!!

template <typename T>
__device__
void traverseTree( T *treeArray_values, int *treeArray_ID, T *box, int pos, int startOfTree, int endOfTree, int number_of_dimensions){
    
    //printf("\n first value: startOfTree+pos*blokDim.y + row %d, row %d", (startOfTree+pos)*blockDim.y+row, row);
    //printf("\n box[0] %d, box[1] %d, box[2] %d, box[3] %d, box[4] %d, box[5] %d, thread %d", box[0], box[1], box[2], box[3], box[4], box[5], threadIdx.x);
    bool flag_not_found_invalid = true;
    int lastLevel = ceil(log2(double(endOfTree+1))-1);
    
    
    if(startOfTree + pos - 1 <= endOfTree){
        //calculate which tree level we are on to know which dimension was sorted
        int level = ceil(log2(double(pos+1))-1);
        int level_of_dimension = level%number_of_dimensions;
        //a mod b = a - floor(a / b) * b
        
        //printf("\n level %d, lastlevel %d,levelOfDimension %d, ID %d, pos %d, startOfTree %d, thrad %d", level, lastLevel,level_of_dimension, treeArray_ID[startOfTree+pos-1], pos, startOfTree, threadIdx.x);
        
        
        
        //check wether invalid encountered, continue search:
        if((treeArray_ID[startOfTree+pos-1] != -1)){
            
            //if node has sorted dimension in box, continue both branches:
            if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] >= box[2*level_of_dimension] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] <= box[2*level_of_dimension+1]){
                
                //check wether the node is inside the box:
                for(int i=0; i<number_of_dimensions; i++){
                    if(i == level_of_dimension) continue;
                    if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                    //entirely inside box for all dimensions
                    }
                    else{
                        //not totally inside box
                        //printf("\n thread %d is changing ID %d of tree starting at %d, exact position: %d", threadIdx.x, treeArray_ID[startOfTree+pos-1], startOfTree, startOfTree+pos-1);
                        treeArray_ID[startOfTree+pos-1] = -2;
                    }
                }
                
                //continue both branches:
                if( ! (level == lastLevel-1 && flag_not_found_invalid==false)){
                    //left child:
                    pos *= 2;
                    traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
                if( ! (level == lastLevel-1 && flag_not_found_invalid==false)){
                    //right child:
                    pos += 1;
                    traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
            //if sorted dimension is larger than box follow branch of smaller child = left child
            else if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level] > box[2*level+1]){
                treeArray_ID[startOfTree+pos-1] = -2;
                if( ! (level == lastLevel-1 && flag_not_found_invalid==false)){
                    //left child:
                    pos *= 2;
                    traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
            //if sorted dimension is smaller than box, follow branch of larger child = right child
            else{
                treeArray_ID[startOfTree+pos-1] = -2;
                if( ! (level == lastLevel-1 && flag_not_found_invalid==false)){
                    //right child:
                    pos += 1;
                    traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
        }
        //if invalid is encountered, all future lastLevel checks don't have to be done
        //flag is set
        else{
            flag_not_found_invalid = false;
        }
            
            
            
            
            
            /*
            if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                
                //inside box
                //printf("\n value: %d , box-min: %d, box-max: %d, ID: %d, thread %d", treeArray_values[startOfTree+number_of_dimensions*(pos-1)+i], box[2*i], box[2*i+1], treeArray_ID[startOfTree+pos-1], threadIdx.x);
            }
            else{
                //printf("\n value: %d , box-min: %d, box-max: %d, ID: %d, thread %d NOOOOOT", treeArray_values[startOfTree+number_of_dimensions*(pos-1)+i], box[2*i], box[2*i+1], treeArray_ID[startOfTree+pos-1], threadIdx.x);
                treeArray_ID[startOfTree+pos-1] = -1;
                
                //left child:
                pos *= 2;
                traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
                
                //right child:
                pos += 1;
                traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);c
            }
            
        }
        
        //Abbruchkriterium:
        //TODO: < oder <= ???
        //TODO: if hier weg
        if(startOfTree+pos < endOfTree){
            
            //left child:
            pos *= 2;
            traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
            
            //right child:
            pos += 1;
            traverseTree(treeArray_values, treeArray_ID, box, pos, startOfTree, endOfTree, number_of_dimensions);
        }*/
    }
}
template <typename T>
__device__
void traverseTreeRecursiveIF( T *treeArray_values, int *treeArray_ID, int *treeArray_results, T *box, int pos, int startOfTree, int endOfTree, int number_of_dimensions){
    
    if(startOfTree + pos - 1 <= endOfTree){
        
        //calculate which tree level we are on to know which dimension was sorted
        int level = ceil(log2(double(pos+1))-1);
        int level_of_dimension = level%number_of_dimensions;
        int lastLevel = ceil(log2(double(endOfTree+1))-1);
        //a mod b = a - floor(a / b) * b
        
        //cout << "global position: " << startOfTree+pos -1 << " und ID: " << treeArray_ID[startOfTree+pos-1] << " level: " << level <<  " levelofDimension: " << level_of_dimension << endl;
        
        
        //check wether invalid encountered, continue search:
        if(treeArray_ID[startOfTree+pos-1] != 0){
            
            //if node has sorted dimension in box, continue both branches:
            //cout << box[2*level_of_dimension] <<  " <= " << treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] << " <= " << box[2*level_of_dimension+1] << endl;
            
            if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] >= box[2*level_of_dimension] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] <= box[2*level_of_dimension+1]){
                bool inside = true;
                //cout << "number of nodes traversed " << number_of_nodes_traversed << endl;
                
                //check wether the node is inside the box:
                for(int i=0; i<number_of_dimensions; i++){
                    if(i == level_of_dimension) continue;
                    //cout << treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] << " >= " << box[2*i] <<  " && " << treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] << " <= " << box[2*i+1] << endl;
                    
                    if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                        //entirely inside box for all dimensions
                        
                    }
                    else{
                        //not totally inside box
                        //printf("\n thread %d is changing ID %d of tree starting at %d, exact position: %d", threadIdx.x, treeArray_ID[startOfTree+pos-1], startOfTree, startOfTree+pos-1);
                        //cout << "not inside \t ID: " << treeArray_ID[startOfTree+pos-1] <<" bei pos: " << startOfTree+pos-1 << " weg! BEIDE BRANCHES" << endl;
                        inside = false;
                    }
                }
                if(inside){
                    treeArray_results[startOfTree+pos-1] = treeArray_ID[startOfTree+pos-1];
                    //cout << "yes inside \t ID: " << treeArray_ID[startOfTree+pos-1] << " BEIDE BRANCHES" << endl;
                }
                
                //continue both branches:
                if(level != lastLevel){
                    //cout << "ID: " << treeArray_ID[startOfTree+pos-1] <<" bei pos: " << startOfTree+pos-1 << " BEIDE BRANCHES" << endl;
                    //left child:
                    pos *= 2;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID,treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                    
                    //right child:
                    pos += 1;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID,treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
                
            }
            //if sorted dimension is larger than box follow branch of smaller child = left child
            else if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] > box[2*level_of_dimension+1]){
                //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " LINKES KIND" << endl;
                //cout << "number of nodes traversed " << number_of_nodes_traversed << endl;
                //left child:
                if(level != lastLevel){
                    pos *= 2;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID,treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
            //if sorted dimension is smaller than box, follow branch of larger child = right child
            else{
                //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " RECHTES KIND" << endl;
                //cout << "number of nodes traversed " << number_of_nodes_traversed << endl;
                if(level != lastLevel){
                    //right child:
                    pos *= 2;
                    pos += 1;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID, treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
        }
    }
}

template <typename T>
__device__
void traverseTreeIterative( T *treeArray_values, int *treeArray_ID, int *treeArray_results, T *box, int *queue, int pos, int startOfTree, int endOfTree, int number_of_dimensions){
    
    int lastLevel = ceil(log2(double(endOfTree+1))-1);
    
    
    queue[startOfTree] = pos;
    int queueFront = 0;
    int queueRear = 1;
    int queueSize = 1;
    int numberOfMightHits = 0;
    
    
    while(queueSize != 0){
        queueSize--;
        pos = queue[queueFront++];
        
        int level = ceil(log2(double(pos+1))-1);
        int level_of_dimension = level%number_of_dimensions;
        
        //cout << "position " << pos-1 << endl;
        
        //if sorted dimension inside box continue with both branches
        if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] >= box[2*level_of_dimension] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] <= box[2*level_of_dimension+1]){
            //put left and right child in queue:
            //cout << "ID: " << treeArray_ID[startOfTree+pos-1] <<" bei pos: " << startOfTree+pos-1 << " BEIDE BRANCHES" << endl;
            
            //nested ifs - bad!
            if(level != lastLevel){
                queue[queueRear++] = pos*2;
                queue[queueRear++] = pos*2+1;
                queueSize+=2;
            }
            
            //and check if node is totally inside box - then right it to results
            //possibility(?) can this be checked after tree traversed? Can an extra thread check this? 2 threads per tree?
            //write pos to array? What about size of array on GPU? Size of tree must be allocated? Is there so much space?
            //per warp needed memory: 3*TreeSize - NO !!! can be put into results :)
            //toCheckIfInside.push_back(pos);
            //these results have to be checked again!!!
            treeArray_results[startOfTree+numberOfMightHits] = pos; //treeArray_ID[startOfTree+pos-1];
            numberOfMightHits++;
            
        }
        //else if sorted dimensions > inside box continue with left child
        else if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] > box[2*level_of_dimension+1]
                && level != lastLevel){
            //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " LINKES KIND" << endl;
            queue[queueRear++] = pos*2;
            queueSize++;
        }
        //else sorted dimension < inside box continue with right child
        else if(level != lastLevel){
            //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " RECHTES KIND" << endl;
            queue[queueRear++] = pos*2+1;
            queueSize++;
        }
    }
    
    //check nodes, that might be inside box:
    for(int j = 0; j<=numberOfMightHits;j++){
        pos = treeArray_results[startOfTree+j];
        treeArray_results[startOfTree+j] = 0;
        
        bool inside = true;
        for(int i=0; i<number_of_dimensions; i++){
            if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                //entirely inside box for all dimensions
            }
            else{
                inside = false;
            }
        }
        if(inside){
            treeArray_results[startOfTree+pos-1] = treeArray_ID[startOfTree+pos-1];
        }
    }
}

template <typename T>
__global__
void insideBox(T *treeArray_values, int *treeArray_ID, int *treeArray_results, T *box, int *queue, int tree_size, int number_of_dimensions){
    
    //for each thread has it's own tree starting here
    //TODO: STARTOFTREE falsch.. ist die gesamte position, ohne berücksichtigung der number_of_dimensions. Die müssen berücksichtigt werden!!!
    int startOfTree = threadIdx.x * tree_size;
    int endOfTree = startOfTree + (tree_size - 1);
    /*for(int i = startOfTree; i<endOfTree; i++ ){
        printf("\n (%d, %d, %d) \t ID: %d,\t startOfTree: %d, \t position of point: %d, thread: %d", treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*i+0], treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*i+1], treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*i+2], treeArray_ID[startOfTree+i], startOfTree*number_of_dimensions, startOfTree*number_of_dimensions+number_of_dimensions*i, threadIdx.x);
    }
    printf("\n threadIdx: %d startOfTree %d, endOfTree %d, row %d, col %d, blockDim %d", threadIdx.x, startOfTree, endOfTree, row, col, blockDim.y);
     */
    //traverseTree(treeArray_values, treeArray_ID, box, 1, startOfTree, endOfTree, number_of_dimensions);
    //traverseTreeRecursiveIF(treeArray_values, treeArray_ID, treeArray_results, box, 1, startOfTree, endOfTree, number_of_dimensions);
    traverseTreeIterative(treeArray_values, treeArray_ID, treeArray_results, box, queue, 1, startOfTree, endOfTree, number_of_dimensions);
}

template <typename T>
void Cuda_class<T>::cudaInsideBox(int number_of_trees, int tree_size, int number_of_dimensions, T *treeArray_values, int *treeArray_ID, int *treeArray_results, T box[], int* queue){
    
    //insideBox<T><<<Anzahl benutzte Blöcke, Anzahl Threads>>> = <<<Anzahl benutzte Blöcke, Anzahl Baeume >>>
    //weil ein Thread == ein Baum
    int max_threads_per_block = 1024;
    int number_of_blocks = (number_of_trees + max_threads_per_block -1) / max_threads_per_block ;
    if(number_of_trees > 1024){number_of_trees = 1024;}
    
    insideBox<T><<<number_of_blocks,number_of_trees>>>(d_treeArray_values, d_treeArray_ID, d_treeArray_results, d_box, d_queue, tree_size, number_of_dimensions);
    //YourKernel<<<dimGrid, dimBlock>>>(d_A,d_B); //Kernel invocation
}

template <typename T>
void Cuda_class<T>::cudaCopyToDevice(int number_of_trees_, int tree_size_, T *treeArray_values, int *treeArray_ID, int* treeArray_results, T box[],int *queue, int number_of_dimensions_){
    number_of_trees = number_of_trees_;
    number_of_dimensions = number_of_dimensions_;
    tree_size = tree_size_;

    hipSetDevice(MYDEVICE);
    std::cout << "number of trees: " << number_of_trees << std::endl;
    std::cout << "tree size: " << tree_size << std::endl;
    std::cout << "number of dimensions: " << number_of_dimensions << std::endl;
    std::cout << "box: " << box[0] << " " << box[1] << " " << box[2] << " " << box[3] << " " << box[4] << " " << box[5] << std::endl;
    //TODO: int ----> num_t
    size_of_forest =  number_of_trees*tree_size*sizeof(int);
    
    //allocate memory
    //TODO: do outside of cudaMain
    hipMalloc(&d_treeArray_values, size_of_forest*number_of_dimensions);
    hipMalloc(&d_treeArray_ID, size_of_forest);
    hipMalloc(&d_treeArray_results, size_of_forest);
    hipMalloc(&d_box, number_of_dimensions*2*sizeof(T));
    hipMalloc(&d_queue, size_of_forest);
    
    //send trees to gpu
    hipMemcpy(d_treeArray_values, treeArray_values, size_of_forest*number_of_dimensions, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_ID, treeArray_ID, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_results, treeArray_results, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_box, box, number_of_dimensions*2*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_queue, queue, size_of_forest, hipMemcpyHostToDevice);
}

template <typename T>
void Cuda_class<T>::cudaCopyToHost(int* treeArray_results){
    
    hipMemcpy(treeArray_results, d_treeArray_results, size_of_forest, hipMemcpyDeviceToHost);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    
    std::cout << "\n Size of forest: " << size_of_forest << std::endl;
    
//    //print out ID's which are in box:
//        for(int i = 0; i< number_of_trees*tree_size; i++){
//            std::cout << "ID: " << treeArray_ID[i]<< std::endl;
//        }
    
    
    //free space
    hipFree(d_treeArray_values);
    hipFree(d_treeArray_ID);
    hipFree(d_treeArray_results);
    hipFree(d_box);

}

template class Cuda_class<int>;
