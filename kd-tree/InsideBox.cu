#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>
#include <iostream>
#include "InsideBox.hpp"


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

template <typename T>
__device__
void traverseTreeRecursiveIF( T *treeArray_values, int *treeArray_ID, int *treeArray_results, T *box, int pos, int startOfTree, int endOfTree, int number_of_dimensions, int startOfBox){
    
    if(startOfTree + pos - 1 <= endOfTree){
        
        //calculate which tree level we are on to know which dimension was sorted
        int level = ceil(log2(double(pos+1))-1);
        int level_of_dimension = level%number_of_dimensions;
        int lastLevel = ceil(log2(double(endOfTree+1))-1);
        //a mod b = a - floor(a / b) * b
        
        //cout << "global position: " << startOfTree+pos -1 << " und ID: " << treeArray_ID[startOfTree+pos-1] << " level: " << level <<  " levelofDimension: " << level_of_dimension << endl;
        
        
        //check wether invalid encountered, continue search:
        if(treeArray_ID[startOfTree+pos-1] != 0){
            
            //if node has sorted dimension in box, continue both branches:
            //cout << box[2*level_of_dimension] <<  " <= " << treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] << " <= " << box[2*level_of_dimension+1] << endl;
            
            if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] >= box[2*level_of_dimension] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] <= box[2*level_of_dimension+1]){
                bool inside = true;
                //cout << "number of nodes traversed " << number_of_nodes_traversed << endl;
                
                //check wether the node is inside the box:
                for(int i=0; i<number_of_dimensions; i++){
                    if(i == level_of_dimension) continue;
                    //cout << treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] << " >= " << box[2*i] <<  " && " << treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] << " <= " << box[2*i+1] << endl;
                    
                    if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                        //entirely inside box for all dimensions
                        
                    }
                    else{
                        //not totally inside box
                        //printf("\n thread %d is changing ID %d of tree starting at %d, exact position: %d", threadIdx.x, treeArray_ID[startOfTree+pos-1], startOfTree, startOfTree+pos-1);
                        //cout << "not inside \t ID: " << treeArray_ID[startOfTree+pos-1] <<" bei pos: " << startOfTree+pos-1 << " weg! BEIDE BRANCHES" << endl;
                        inside = false;
                    }
                }
                if(inside){
                    treeArray_results[startOfTree+pos-1] = treeArray_ID[startOfTree+pos-1];
                    //cout << "yes inside \t ID: " << treeArray_ID[startOfTree+pos-1] << " BEIDE BRANCHES" << endl;
                }
                
                //continue both branches:
                if(level != lastLevel){
                    //cout << "ID: " << treeArray_ID[startOfTree+pos-1] <<" bei pos: " << startOfTree+pos-1 << " BEIDE BRANCHES" << endl;
                    //left child:
                    pos *= 2;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID,treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                    
                    //right child:
                    pos += 1;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID,treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
                
            }
            //if sorted dimension is larger than box follow branch of smaller child = left child
            else if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] > box[2*level_of_dimension+1]){
                //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " LINKES KIND" << endl;
                //cout << "number of nodes traversed " << number_of_nodes_traversed << endl;
                //left child:
                if(level != lastLevel){
                    pos *= 2;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID,treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
            //if sorted dimension is smaller than box, follow branch of larger child = right child
            else{
                //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " RECHTES KIND" << endl;
                //cout << "number of nodes traversed " << number_of_nodes_traversed << endl;
                if(level != lastLevel){
                    //right child:
                    pos *= 2;
                    pos += 1;
                    traverseTreeRecursiveIF(treeArray_values, treeArray_ID, treeArray_results, box, pos, startOfTree, endOfTree, number_of_dimensions);
                }
            }
        }
    }
}

template <typename T>
__device__
void traverseTreeIterative( T *treeArray_values, int *treeArray_ID, int *treeArray_results, T *box, int *queue, int pos, int startOfTree, int endOfTree, int number_of_dimensions, int startOfBox){
    
    int lastLevel = ceil(log2(double(endOfTree+1))-1);
    
    queue[startOfTree] = pos;
    int queueFront = startOfTree;
    int queueRear = startOfTree+1;
    int queueSize = 1;
    int numberOfMightHits = 0;
    
    
    while(queueSize != 0){
        queueSize--;
        pos = queue[queueFront++];
        
        int level = ceil(log2(double(pos+1))-1);
        int level_of_dimension = level%number_of_dimensions;
        
        //cout << "position " << pos-1 << endl;
        
        //if sorted dimension inside box continue with both branches
        if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] >= box[2*level_of_dimension] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] <= box[2*level_of_dimension+1]){
            //put left and right child in queue:
            //cout << "ID: " << treeArray_ID[startOfTree+pos-1] <<" bei pos: " << startOfTree+pos-1 << " BEIDE BRANCHES" << endl;
            
            //nested ifs - bad!
            if(level != lastLevel){
                queue[queueRear++] = pos*2;
                queue[queueRear++] = pos*2+1;
                queueSize+=2;
            }
            
            //and check if node is totally inside box - then right it to results
            //possibility(?) can this be checked after tree traversed? Can an extra thread check this? 2 threads per tree?
            //write pos to array? What about size of array on GPU? Size of tree must be allocated? Is there so much space?
            //per warp needed memory: 3*TreeSize - NO !!! can be put into results :)
            //toCheckIfInside.push_back(pos);
            //these results have to be checked again!!!
            treeArray_results[startOfTree+numberOfMightHits] = pos; //treeArray_ID[startOfTree+pos-1];
            numberOfMightHits++;
            
        }
        //else if sorted dimensions > inside box continue with left child
        else if(treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+level_of_dimension] > box[2*level_of_dimension+1]
                && level != lastLevel){
            //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " LINKES KIND" << endl;
            queue[queueRear++] = pos*2;
            queueSize++;
        }
        //else sorted dimension < inside box continue with right child
        else if(level != lastLevel){
            //cout << "ID: " << treeArray_ID[startOfTree+pos-1] << " bei pos: " << startOfTree+pos-1 << " RECHTES KIND" << endl;
            queue[queueRear++] = pos*2+1;
            queueSize++;
        }
    }
    
    //check nodes, that might be inside box:
    for(int j = 0; j<=numberOfMightHits;j++){
        pos = treeArray_results[startOfTree+j];
        treeArray_results[startOfTree+j] = 0;
        
        bool inside = true;
        for(int i=0; i<number_of_dimensions; i++){
            if( treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] >= box[2*i] && treeArray_values[startOfTree*number_of_dimensions+number_of_dimensions*(pos-1)+i] <= box[2*i+1] ){
                //entirely inside box for all dimensions
            }
            else{
                inside = false;
            }
        }
        if(inside){
            treeArray_results[startOfTree+pos-1] = treeArray_ID[startOfTree+pos-1];
        }
    }
}

template <typename T>
__global__
void insideBox(T *treeArray_values, int *treeArray_ID, int *treeArray_results, T *box, int *queue, int tree_size, int number_of_dimensions){
    
    //__shared__ int s[]
    //for each thread has it's own tree starting here
    //TODO: STARTOFTREE falsch.. ist die gesamte position, ohne berücksichtigung der number_of_dimensions. Die müssen berücksichtigt werden!!!
    int startOfTree = threadIdx.x * tree_size;
    int endOfTree = startOfTree + (tree_size - 1);
    int startOfBox = (blockIdx.x*blockDim.x + threadIdx.x)*number_of_dimensions*2;
    int startOfResults = (blockIdx.x*blockDim.x + threadIdx.x)*tree_size;
    printf("\n Index for large arrays -  Blockdim.x: %d, blockIdx.x %d, threadIdx.x %d, blockIdx.x*blockDim.x + threadIdx.x %d", blockDim.x , blockIdx.x, threadIdx.x, blockIdx.x*blockDim.x + threadIdx.x);
    
<<<<<<< HEAD
    //traverseTreeRecursiveIF(treeArray_values, treeArray_ID, treeArray_results, box, 1, startOfTree, endOfTree, number_of_dimensions, startOfBox);
    //traverseTreeIterative(treeArray_values, treeArray_ID, treeArray_results, box, queue, 1, startOfTree, endOfTree, number_of_dimensions, startOfBox);
=======
    //traverseTreeRecursiveIF(treeArray_values, treeArray_ID, treeArray_results, box, 1, startOfTree, endOfTree, number_of_dimensions);
    traverseTreeIterative(treeArray_values, treeArray_ID, treeArray_results, box, queue, 1, startOfTree, endOfTree, number_of_dimensions);
>>>>>>> parent of 5f29d17... one hit for every tree inside box
}

template <typename T>
void Cuda_class<T>::cudaInsideBox(int number_of_trees, int tree_size, int number_of_dimensions, T *treeArray_values, int *treeArray_ID, int *treeArray_results, T box[], int* queue, int numberOfHits){
    //number_of_warps = numberOfHits
    //numberOfBlocks = numberOfHits/32, because 32 = 1 warp
    
    
    //insideBox<T><<<Anzahl benutzte Blöcke, Anzahl Threads>>> = <<<Anzahl benutzte Blöcke, Anzahl Baeume >>>
<<<<<<< HEAD
    //weil ein Thread == ein Baum, ein warp == eine box
    int warp_size = 32;
    int number_of_blocks = (numberOfHits+32-1)/32;
    
    std::cout << "Number of Blocks: " << number_of_blocks << "\t number of Threads: " << numberOfHits*warp_size << std::endl;
    
    insideBox<T><<<number_of_blocks,1024>>>(d_treeArray_values, d_treeArray_ID, d_treeArray_results, d_box, d_queue, tree_size, number_of_dimensions);
=======
    //weil ein Thread == ein Baum
    int max_threads_per_block = 1024;
    int number_of_blocks = (number_of_trees + max_threads_per_block -1) / max_threads_per_block ;
    if(number_of_trees > 1024){number_of_trees = 1024;}
    
    insideBox<T><<<number_of_blocks,number_of_trees>>>(d_treeArray_values, d_treeArray_ID, d_treeArray_results, d_box, d_queue, tree_size, number_of_dimensions);
>>>>>>> parent of 5f29d17... one hit for every tree inside box
    //YourKernel<<<dimGrid, dimBlock>>>(d_A,d_B); //Kernel invocation
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error after Kernel: %s\n", hipGetErrorString(err));
}

template <typename T>
void Cuda_class<T>::cudaCopyToDevice(int number_of_trees_, int tree_size_, T *treeArray_values, int *treeArray_ID, int* treeArray_results, T box[], int *queue, int number_of_dimensions_, int numberOfHits){
    number_of_trees = number_of_trees_;
    number_of_dimensions = number_of_dimensions_;
    tree_size = tree_size_;
    size_of_forest =  number_of_trees*tree_size;
    
    hipSetDevice(MYDEVICE);
    std::cout << "number of trees: " << number_of_trees << std::endl;
    std::cout << "tree size: " << tree_size << std::endl;
    std::cout << "number of dimensions: " << number_of_dimensions << std::endl;
    std::cout << "size of treeArray_values " << size_of_forest*number_of_dimensions << std::endl;
    std::cout << "size of treeArray_ID " << size_of_forest << std::endl;
    std::cout << "size of treeArray_results " << size_of_forest*numberOfHits << std::endl;
    std::cout << "size of box " << number_of_dimensions*2*numberOfHits << std::endl;
    std::cout << "size of queue " << size_of_forest << std::endl;
    
    //std::cout << "box: " << box[0] << " " << box[1] << " " << box[2] << " " << box[3] << " " << box[4] << " " << box[5] << std::endl;
    //TODO: int ----> num_t
    
    //allocate memory
<<<<<<< HEAD
    hipError_t errortreeArray = hipMalloc(&d_treeArray_values, size_of_forest*number_of_dimensions*sizeof(T));
    hipError_t errortreeID = hipMalloc(&d_treeArray_ID, size_of_forest*sizeof(int));
    hipError_t errorResults = hipMalloc(&d_treeArray_results, size_of_forest*numberOfHits*sizeof(int));
    hipError_t errorBox = hipMalloc(&d_box, number_of_dimensions*2*numberOfHits*sizeof(T));
    hipError_t errorQueue = hipMalloc(&d_queue, size_of_forest*numberOfHits*sizeof(T));
    if (errorResults != hipErrorOutOfMemory)
        printf("Error in allocating memory cudaResult: %s\n", hipGetErrorString(errorResults));
    if (errortreeArray != hipErrorOutOfMemory)
        printf("Error in allocating memory cudatreeArray: %s\n", hipGetErrorString(errortreeArray));
    if (errortreeID != hipErrorOutOfMemory)
        printf("Error in allocating memory treeID: %s\n", hipGetErrorString(errortreeID));
    if (errorBox != hipErrorOutOfMemory)
        printf("Error in allocating memory box: %s\n", hipGetErrorString(errorBox));
    if (errorQueue != hipErrorOutOfMemory)
        printf("Error in allocating memory queue: %s\n", hipGetErrorString(errorQueue));
    
=======
    //TODO: do outside of cudaMain
    hipMalloc(&d_treeArray_values, size_of_forest*number_of_dimensions);
    hipMalloc(&d_treeArray_ID, size_of_forest);
    hipMalloc(&d_treeArray_results, size_of_forest);
    hipMalloc(&d_box, number_of_dimensions*2*sizeof(T));
    hipMalloc(&d_queue, size_of_forest);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error in allocating memory: %s\n", hipGetErrorString(err));
>>>>>>> parent of 5f29d17... one hit for every tree inside box
    
    //send trees to gpu
    hipMemcpy(d_treeArray_values, treeArray_values, size_of_forest*number_of_dimensions*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_ID, treeArray_ID, size_of_forest*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_results, treeArray_results, size_of_forest*numberOfHits*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_box, box, number_of_dimensions*2*numberOfHits*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_queue, queue, size_of_forest*numberOfHits*sizeof(T), hipMemcpyHostToDevice);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error in sending stuff: %s\n", hipGetErrorString(err));
}

template <typename T>
void Cuda_class<T>::cudaCopyToHost(int* treeArray_results, int numberOfHits){
    
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error before copying back to host: %s\n", hipGetErrorString(err));
    
    hipMemcpy(treeArray_results, d_treeArray_results, size_of_forest*numberOfHits*sizeof(int), hipMemcpyDeviceToHost);
    
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error after copying back to host: %s\n", hipGetErrorString(err));
    
    std::cout << "\n Size of forest: " << size_of_forest << std::endl;
    
    //    //print out ID's which are in box:
    //        for(int i = 0; i< number_of_trees*tree_size; i++){
    //            std::cout << "ID: " << treeArray_ID[i]<< std::endl;
    //        }
    
    
    //free space
    hipFree(d_treeArray_values);
    hipFree(d_treeArray_ID);
    hipFree(d_treeArray_results);
    hipFree(d_box);
    hipFree(d_queue);
    std::cout <<"Freed cuda stuff " << std::endl;
    
}

template class Cuda_class<int>;
template class Cuda_class<double>;