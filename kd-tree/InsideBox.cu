#include "hip/hip_runtime.h"
//
//  InsideBox.cu
//  
//
//  Created by Ann-Christine Vossberg on 6/3/15.
//
//

#include <stdio.h>
#include <iostream>
//#include "cuPrintf.cu" braucht es?
#include "InsideBox.hpp"


// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);


//think about: threadIDx.y!! index s.d. threadIDx.y die treeArray_y bearbeitet -
//for this would have to change treeArray_x & treeArray_y etc --> tree..
//no nested if's no recursive.. -
__global__
void insideBox( int *treeArray_x, int *treeArray_y, int *treeArray_z, int *treeArray_ID, int *box)
{
    //TODO: change to get warpSize
    int warpSize = 32;
    int warpIdx = threadIdx.x / warpSize;
    int i = warpIdx;
    int j = threadIdx.x % 32; //=0 bis 32
    int treeSize = warpSize-1;
    int index = i*treeSize+j;
    
    
    
    //evaluate with another if(which is the next, then you exit from the if.. and you follow the next
    if( ((treeArray_x[index] >= box[0] && treeArray_x[index] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[index] >= box[2] && treeArray_y[index] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[index] >= box[4] && treeArray_z[index] <= box[5]) || (box[4] == 0 && box[5] == 0))){
        //inside box
        
    }
        else{
            //printf("not inside box");
        treeArray_ID[index] = -1;
    }
}

/*__device__
void traverseTree(int *treeArray_x, int *treeArray_y, int *treeArray_z, int *treeArray_ID, int *box, int pos, int startOfTree, int endOfTree){
    
    //check if inside box:
    if( ((treeArray_x[pos] >= box[0] && treeArray_x[pos] <= box[1]) || (box[0] == 0 && box[1] == 0))  && ((treeArray_y[pos] >= box[2] && treeArray_y[pos] <= box[3]) || (box[2] == 0 && box[3] == 0)) && ((treeArray_z[pos] >= box[4] && treeArray_z[pos] <= box[5]) || (box[4] == 0 && box[5] == 0))){
        //inside box
        printf("I am thread nr. %d", threadIdx.x);
    }
    else{
        printf("not inside box");
        treeArray_ID[pos] = -1;
    }
    
    //Abbruchkriterium:
    //TODO: < oder <= ???
    if(pos <= endOfTree){
        
       //left child:
        pos = pos + pos % startOfTree; //caution: nicht *= 2; weil
       traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, pos, startOfTree, endOfTree);
        
       //right child:
       pos+=1;
       traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, box, pos, startOfTree, endOfTree);
    }
}


//Each thread starts at Node 0 of it's "own" tree. Traverses tree and changes treeArray_ID to -1, if not inside box.
__global__
void Insidebox(int *treeArray_x, int *treeArray_y, int *treeArray_z, int *treeArray_ID, int *box, int tree_size){
    
    //for each thread has it's own tree starting here (+1, because of 2*i, 2*i+1 sonst = 0)
    int startOfTree = threadIdx.x * tree_size + 1; //1, 32, 64, ...
    int endOfTree = startOfTree + tree_size;
    
    traverseTree(treeArray_x, treeArray_y, treeArray_z, treeArray_ID, *box, startOfTree);

}
*/


//TODO: template
void cudaMain(int number_of_trees, int tree_size, int treeArray_x[], int treeArray_y[], int treeArray_z[], int treeArray_ID[], int box[]){
    
    hipSetDevice(MYDEVICE);
    std::cout << "number of trees: " << number_of_trees << std::endl;
    std::cout << "tree size: " << tree_size << std::endl;
    
    //TODO: int ----> num_t
    int size_of_forest = number_of_trees*tree_size*sizeof(int);
    int *d_treeArray_x;
    int *d_treeArray_y;
    int *d_treeArray_z;
    int *d_treeArray_ID;
    int *d_box;
    
    
    //allocate memory
    hipMalloc(&d_treeArray_x, size_of_forest);
    hipMalloc(&d_treeArray_y, size_of_forest);
    hipMalloc(&d_treeArray_z, size_of_forest);
    hipMalloc(&d_treeArray_ID, size_of_forest);
    //TODO: generic
    hipMalloc(&d_box, 6*sizeof(int));
    
    //send trees to gpu
    hipMemcpy(d_treeArray_x, treeArray_x, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_y, treeArray_y, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_z, treeArray_z, size_of_forest, hipMemcpyHostToDevice);
    hipMemcpy(d_treeArray_ID, treeArray_ID, size_of_forest, hipMemcpyHostToDevice);
    //TODO: generic
    hipMemcpy(d_box, box, 6*sizeof(int), hipMemcpyHostToDevice);
    
    
    //search forest for points inside box_dimensions - returns all treeArray_ID's which are inside box - rest are filled with -1
    insideBox<<<1,1024>>>(d_treeArray_x, d_treeArray_y, d_treeArray_z, d_treeArray_ID, d_box);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    
    hipMemcpy(treeArray_ID, d_treeArray_ID, size_of_forest, hipMemcpyDeviceToHost);
    
    
    std::cout << "Size of forest: " << size_of_forest << std::endl;
    //print out ID's which are in box:
    for(int i = 0; i< number_of_trees*tree_size; i++){
        std::cout << "ID: " << treeArray_ID[i]<< std::endl;
    }
    
    
    //free space
    hipFree(d_treeArray_x);
    hipFree(d_treeArray_y);
    hipFree(d_treeArray_z);
    hipFree(d_treeArray_ID);
    hipFree(d_box);

}
