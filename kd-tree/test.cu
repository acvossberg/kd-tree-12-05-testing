#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "test.hpp"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel( int *d_a, int numBlocks )
{
    int i = numBlocks*blockIdx.x + threadIdx.x;
    d_a[i] = blockIdx.x + threadIdx.x;
    
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void test( )
{
    hipSetDevice(MYDEVICE);
    // pointer for host memory
    int *h_a;
    
    // pointer for device memory
    int *d_a;
    
    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;
    
    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc((void **)&d_a, memSize);
    
    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid( 8 );
    dim3 dimBlock( 8 );
    myFirstKernel<<< 8 , 8 >>>(d_a, numBlocks);
    
    // block until the device has completed
    hipDeviceSynchronize();
    
    // check if kernel execution generated an error
    checkCUDAError("kernel execution");
    
    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost );
    
    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");
    
    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i < numBlocks  ; i++)
    {
        for (int j = 0; j < numThreadsPerBlock  ; j++)
        {
            assert(h_a[i * numThreadsPerBlock + j] == i + j);
        }
    }
    
    // free device memory
    hipFree(d_a);
    
    // free host memory
    free(h_a);
    
    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");
    
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
